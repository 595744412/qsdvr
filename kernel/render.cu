#include "hip/hip_runtime.h"
#include <render.h>

const unsigned int dataCount = 32;

__device__ __constant__ const float g_SHFactor[] =
    {
        0.28209479177387814347403972578039f,
        0.48860251190291992158638462283835f,
        0.48860251190291992158638462283835f,
        0.48860251190291992158638462283835f,
        1.0925484305920790705433857058027f,
        1.0925484305920790705433857058027f,
        0.31539156525252000603089369029571f,
        1.0925484305920790705433857058027f,
        0.54627421529603953527169285290135f};

__device__ __forceinline__ vec3f GetSH3Irradiance(vec3f v, vec3f coef[9])
{
    float x_4 = v.x * v.z;
    float zz = v.z * v.z;
    float xx = v.x * v.x;
    float x_5 = v.x * v.y;
    float x_6 = 2.0 * v.y * v.y - zz - xx;
    float x_7 = v.y * v.z;
    float x_8 = zz - xx;
    float x =
        g_SHFactor[0] * coef[0].x +
        g_SHFactor[1] * coef[1].x * v.x +
        g_SHFactor[2] * coef[2].x * v.y +
        g_SHFactor[3] * coef[3].x * v.z +
        g_SHFactor[4] * coef[4].x * x_4 +
        g_SHFactor[5] * coef[5].x * x_5 +
        g_SHFactor[6] * coef[6].x * x_6 +
        g_SHFactor[7] * coef[7].x * x_7 +
        g_SHFactor[8] * coef[8].x * x_8;

    float y =
        g_SHFactor[0] * coef[0].y +
        g_SHFactor[1] * coef[1].y * v.x +
        g_SHFactor[2] * coef[2].y * v.y +
        g_SHFactor[3] * coef[3].y * v.z +
        g_SHFactor[4] * coef[4].y * x_4 +
        g_SHFactor[5] * coef[5].y * x_5 +
        g_SHFactor[6] * coef[6].y * x_6 +
        g_SHFactor[7] * coef[7].y * x_7 +
        g_SHFactor[8] * coef[8].y * x_8;

    float z =
        g_SHFactor[0] * coef[0].z +
        g_SHFactor[1] * coef[1].z * v.x +
        g_SHFactor[2] * coef[2].z * v.y +
        g_SHFactor[3] * coef[3].z * v.z +
        g_SHFactor[4] * coef[4].z * x_4 +
        g_SHFactor[5] * coef[5].z * x_5 +
        g_SHFactor[6] * coef[6].z * x_6 +
        g_SHFactor[7] * coef[7].z * x_7 +
        g_SHFactor[8] * coef[8].z * x_8;
    return vec3f{x, y, z};
}

__device__ __forceinline__ void GetSH3IrradianceBackward(vec3f out_grad, vec3f v, vec3f coef[9], vec3f &v_grad, vec3f coef_grad[9])
{
    float x_4 = v.x * v.z;
    float zz = v.z * v.z;
    float xx = v.x * v.x;
    float x_5 = v.x * v.y;
    float x_6 = 2.0 * v.y * v.y - zz - xx;
    float x_7 = v.y * v.z;
    float x_8 = zz - xx;
    float gc[8] = {
        g_SHFactor[1] * (coef[1].x * out_grad.x + coef[1].y * out_grad.y + coef[1].z * out_grad.z),
        g_SHFactor[2] * (coef[2].x * out_grad.x + coef[2].y * out_grad.y + coef[2].z * out_grad.z),
        g_SHFactor[3] * (coef[3].x * out_grad.x + coef[3].y * out_grad.y + coef[3].z * out_grad.z),
        g_SHFactor[4] * (coef[4].x * out_grad.x + coef[4].y * out_grad.y + coef[4].z * out_grad.z),
        g_SHFactor[5] * (coef[5].x * out_grad.x + coef[5].y * out_grad.y + coef[5].z * out_grad.z),
        g_SHFactor[6] * (coef[6].x * out_grad.x + coef[6].y * out_grad.y + coef[6].z * out_grad.z),
        g_SHFactor[7] * (coef[7].x * out_grad.x + coef[7].y * out_grad.y + coef[7].z * out_grad.z),
        g_SHFactor[8] * (coef[8].x * out_grad.x + coef[8].y * out_grad.y + coef[8].z * out_grad.z)};
    v_grad.x = gc[0] + v.z * gc[3] + v.y * gc[4] - 2 * v.x * (gc[5] + gc[7]);
    v_grad.y = gc[1] + v.z * gc[6] + v.x * gc[4] + 4 * v.y * gc[5];
    v_grad.z = gc[2] + v.x * gc[3] + v.y * gc[6] + 2 * v.z * (gc[7] - gc[5]);
    coef_grad[0] = out_grad * g_SHFactor[0];
    coef_grad[1] = out_grad * (g_SHFactor[1] * v.x);
    coef_grad[2] = out_grad * (g_SHFactor[2] * v.y);
    coef_grad[3] = out_grad * (g_SHFactor[3] * v.z);
    coef_grad[4] = out_grad * (g_SHFactor[4] * x_4);
    coef_grad[5] = out_grad * (g_SHFactor[5] * x_5);
    coef_grad[6] = out_grad * (g_SHFactor[6] * x_6);
    coef_grad[7] = out_grad * (g_SHFactor[7] * x_7);
    coef_grad[8] = out_grad * (g_SHFactor[8] * x_8);
}

__device__ __forceinline__ float Interpolation1D(float a, float b, float x)
{
    return a * (1 - x) + b * x;
}

__global__ void GridInterpolationForwardKernel(float *out, float *dataGrid, vec3f *pointList, int *indexList, const unsigned int theardCount, unsigned int reso)
{
    const unsigned int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id >= theardCount)
    {
        return;
    }
    const int nid = id % dataCount;
    const int pid = id / dataCount;
    vec3f p = pointList[pid];
    int i000 = indexList[pid] * dataCount + nid;
    int i010 = i000 + reso * dataCount;
    int i100 = i000 + reso * reso * dataCount;
    int i110 = i100 + reso * dataCount;
    float a00 = Interpolation1D(dataGrid[i000], dataGrid[i000 + 1], p.x);
    float a01 = Interpolation1D(dataGrid[i010], dataGrid[i010 + 1], p.x);
    float a0 = Interpolation1D(a00, a01, p.y);
    float a10 = Interpolation1D(dataGrid[i100], dataGrid[i100 + 1], p.x);
    float a11 = Interpolation1D(dataGrid[i110], dataGrid[i110 + 1], p.x);
    float a1 = Interpolation1D(a10, a11, p.y);
    out[id] = Interpolation1D(a0, a1, p.z);
}

__global__ void ShaderForwardKernel(vec3f *out, vec3f *normalList, vec3f *viewDirList, float *dataList, const unsigned int theardCount)
{
    const unsigned int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id >= theardCount)
    {
        return;
    }
    vec3f normal = normalList[id];
    vec3f view = viewDirList[id];
    unsigned int offset = id * 32;
    vec3f specular[9];
#pragma unroll
    for (int i = 0; i < 9; i += 1)
    {
        specular[i] = {dataList[offset + i], dataList[offset + i + 1], dataList[offset + i + 2]};
    }
    vec3f diffuse(dataList[offset + 27], dataList[offset + 28], dataList[offset + 29]);
    float metallic = dataList[offset + 30];
    float ao = dataList[offset + 31];
    float vdotn = view.x * normal.x + view.y * normal.y + view.z * normal.z;
    vec3f reflect = view - normal * (2.0f * vdotn);
    vec3f color = diffuse * (1.0f - metallic) + GetSH3Irradiance(reflect, specular) * (metallic + (1.0f - metallic) * 0.04f);
    out[id] = color * ao;
}

__global__ void RayAggregateForwardKernel(vec3f *out, vec3f *rgbList, float *sdfList, RayInfo *rayList, float logisticCoef, const unsigned int theardCount)
{
    const unsigned int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id >= theardCount)
    {
        return;
    }
    RayInfo info = rayList[id];
    const unsigned int sdfOffset = info.sdfOffset;
    const unsigned int renderOffset = info.renderOffset;
    vec3f color;
    if (info.sdfCount == 0)
    {
        color = {0.0f, 0.0f, 0.0f};
    }
    else
    {

        float exp_SDF_i = expf(-logisticCoef * sdfList[sdfOffset]);
        float T = 1.0f;
        for (unsigned int i = 1; i < info.sdfCount; i += 1)
        {
            float exp_SDF_i_1 = expf(-logisticCoef * sdfList[sdfOffset + i]);
            float alpha = fmaxf((exp_SDF_i_1 - exp_SDF_i) / (1.0f + exp_SDF_i_1), 0.0f);
            color = color + rgbList[renderOffset + i] * alpha * T;
            exp_SDF_i = exp_SDF_i_1;
            T *= (1 - alpha);
        }
    }
    __syncthreads();
    out[id] = color;
}

void GridInterpolationForward(Tensor &dataGrid, Tensor &pointList, Tensor &indexList, Tensor &out, unsigned int reso)
{
    const unsigned int theardCount = dataCount * pointList.size(0);
    const unsigned int blockSize = commonBlockSize;
    const unsigned int gridSize = GetGridSize(blockSize, theardCount);
    GridInterpolationForwardKernel<<<gridSize, blockSize>>>(out.data<float>(), dataGrid.data<float>(), (vec3f *)pointList.data<float>(), indexList.data<int>(), theardCount, reso);
}

void ShaderForward(Tensor &out, Tensor &normalList, Tensor &viewDirList, Tensor &dataList)
{
    const unsigned int theardCount = normalList.size(0);
    const unsigned int blockSize = commonBlockSize;
    const unsigned int gridSize = GetGridSize(blockSize, theardCount);
    ShaderForwardKernel<<<gridSize, blockSize>>>((vec3f *)out.data<float>(), (vec3f *)normalList.data<float>(), (vec3f *)viewDirList.data<float>(), dataList.data<float>(), theardCount);
}

void RayAggregateForward(Tensor &out, Tensor &rgbList, Tensor &sdfList, Tensor &rayList, float logisticCoef)
{
    const unsigned int theardCount = rayList.size(0);
    const unsigned int blockSize = commonBlockSize;
    const unsigned int gridSize = GetGridSize(blockSize, theardCount);
    RayAggregateForwardKernel<<<gridSize, blockSize>>>((vec3f *)out.data<float>(), (vec3f *)rgbList.data<float>(), sdfList.data<float>(), (RayInfo *)rayList.data<int>(), logisticCoef, theardCount);
}
