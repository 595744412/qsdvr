#include "hip/hip_runtime.h"
#include <surface.h>

__global__ void LayerToGridForwardKernel(float *out, float *xLayer_a0Cuda, float *yLayer_a1Cuda,
                                         float *zLayer_a2Cuda, float *xLayer_a3Cuda, float *yLayer_a4Cuda,
                                         float *zLayer_a5Cuda, float *xLayer_a6Cuda, float *yLayer_a6Cuda,
                                         float *zLayer_a6Cuda, float offset_a6, const unsigned int theardCount, unsigned int reso)
{
    unsigned int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id >= theardCount)
    {
        return;
    }
    const int x = id % reso;
    const int y = (id / reso) % reso;
    const int z = id / (reso * reso);
    id *= 7;
    out[id] = xLayer_a0Cuda[x];
    out[id + 1] = yLayer_a1Cuda[y];
    out[id + 2] = zLayer_a2Cuda[z];
    out[id + 3] = xLayer_a3Cuda[x];
    out[id + 4] = yLayer_a4Cuda[y];
    out[id + 5] = zLayer_a5Cuda[z];
    out[id + 6] = xLayer_a6Cuda[x] + yLayer_a6Cuda[y] + zLayer_a6Cuda[z] + offset_a6;
}

__global__ void SampleSDFForwardKernel(float *out, float *sdfGrid, vec3f *pointList, int *indexList, const unsigned int theardCount)
{
    unsigned int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id >= theardCount)
    {
        return;
    }
    int index = indexList[id];
    vec3f p = pointList[id];
    index *= 7;
    float a0 = sdfGrid[index] * p.x;
    float a1 = sdfGrid[index + 1] * p.y;
    float a2 = sdfGrid[index + 2] * p.z;
    float a3 = sdfGrid[index + 3];
    float a4 = sdfGrid[index + 4];
    float a5 = sdfGrid[index + 5];
    float a6 = sdfGrid[index + 6];
    out[id] = (a0 * p.x + a1 * p.y + a2 * p.z + a3 * p.x + a4 * p.y + a5 * p.z + a6) / vec3f(2 * a0 + a3, 2 * a1 + a4, 2 * a2 + a5).Norm();
}

__global__ void SampleNormalForwardKernel(vec3f *out, float *sdfGrid, vec3f *pointList, int *indexList, const unsigned int theardCount)
{
    unsigned int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id >= theardCount)
    {
        return;
    }
    int index = indexList[id];
    vec3f p = pointList[id];
    index *= 7;
    out[id] = vec3f(2 * sdfGrid[index] * p.x + sdfGrid[index + 3], 2 * sdfGrid[index + 1] * p.y + sdfGrid[index + 4], 2 * sdfGrid[index + 2] * p.z + sdfGrid[index + 5]).Normalize();
}

void LayerToGridForward(Tensor &out, Tensor &xLayer, Tensor &yLayer, Tensor &zLayer, Tensor &offset)
{
    unsigned int reso = xLayer.size(0);
    float *xLayer_a0 = xLayer.data<float>();
    float *yLayer_a1 = yLayer.data<float>();
    float *zLayer_a2 = zLayer.data<float>();
    float *offset_ = offset.data<float>();
    float *xLayer_a3 = new float[reso];
    float *yLayer_a4 = new float[reso];
    float *zLayer_a5 = new float[reso];
    float *xLayer_a6 = new float[reso];
    float *yLayer_a6 = new float[reso];
    float *zLayer_a6 = new float[reso];
    xLayer_a3[0] = offset_[0];
    yLayer_a4[0] = offset_[1];
    zLayer_a5[0] = offset_[2];
    xLayer_a6[0] = 0;
    yLayer_a6[0] = 0;
    zLayer_a6[0] = 0;
    for (unsigned int i = 1; i < reso; i += 1)
    {
        xLayer_a3[i] = 2 * xLayer_a0[i - 1] + 2 * xLayer_a0[i] + xLayer_a3[i - 1];
        yLayer_a4[i] = 2 * yLayer_a1[i - 1] + 2 * yLayer_a1[i] + yLayer_a4[i - 1];
        zLayer_a5[i] = 2 * zLayer_a2[i - 1] + 2 * zLayer_a2[i] + zLayer_a5[i - 1];
    }
    for (unsigned int i = 1; i < reso; i += 1)
    {
        xLayer_a6[i] = 3 * xLayer_a0[i - 1] + xLayer_a0[i] + 2 * xLayer_a3[i - 1] + xLayer_a6[i - 1];
        yLayer_a6[i] = 3 * yLayer_a1[i - 1] + yLayer_a1[i] + 2 * yLayer_a4[i - 1] + yLayer_a6[i - 1];
        zLayer_a6[i] = 3 * zLayer_a2[i - 1] + zLayer_a2[i] + 2 * zLayer_a5[i - 1] + zLayer_a6[i - 1];
    }
    float *xLayer_a0Cuda;
    float *yLayer_a1Cuda;
    float *zLayer_a2Cuda;
    float *xLayer_a3Cuda;
    float *yLayer_a4Cuda;
    float *zLayer_a5Cuda;
    float *xLayer_a6Cuda;
    float *yLayer_a6Cuda;
    float *zLayer_a6Cuda;
    size_t l = reso * sizeof(float);
    hipMalloc((void **)&xLayer_a0Cuda, l);
    hipMalloc((void **)&yLayer_a1Cuda, l);
    hipMalloc((void **)&zLayer_a2Cuda, l);
    hipMalloc((void **)&xLayer_a3Cuda, l);
    hipMalloc((void **)&yLayer_a4Cuda, l);
    hipMalloc((void **)&zLayer_a5Cuda, l);
    hipMalloc((void **)&xLayer_a6Cuda, l);
    hipMalloc((void **)&yLayer_a6Cuda, l);
    hipMalloc((void **)&zLayer_a6Cuda, l);
    hipMemcpy(xLayer_a0Cuda, xLayer_a0, l, hipMemcpyHostToDevice);
    hipMemcpy(yLayer_a1Cuda, yLayer_a1, l, hipMemcpyHostToDevice);
    hipMemcpy(zLayer_a2Cuda, zLayer_a2, l, hipMemcpyHostToDevice);
    hipMemcpy(xLayer_a3Cuda, xLayer_a3, l, hipMemcpyHostToDevice);
    hipMemcpy(yLayer_a4Cuda, yLayer_a4, l, hipMemcpyHostToDevice);
    hipMemcpy(zLayer_a5Cuda, zLayer_a5, l, hipMemcpyHostToDevice);
    hipMemcpy(xLayer_a6Cuda, xLayer_a6, l, hipMemcpyHostToDevice);
    hipMemcpy(yLayer_a6Cuda, yLayer_a6, l, hipMemcpyHostToDevice);
    hipMemcpy(zLayer_a6Cuda, zLayer_a6, l, hipMemcpyHostToDevice);
    const unsigned int theardCount = reso * reso * reso;
    const unsigned int blockSize = commonBlockSize;
    const unsigned int gridSize = GetGridSize(blockSize, theardCount);
    LayerToGridForwardKernel<<<gridSize, blockSize>>>(out.data<float>(), xLayer_a0Cuda, yLayer_a1Cuda,
                                                      zLayer_a2Cuda, xLayer_a3Cuda, yLayer_a4Cuda,
                                                      zLayer_a5Cuda, xLayer_a6Cuda, yLayer_a6Cuda,
                                                      zLayer_a6Cuda, offset_[3], theardCount, reso);
    hipFree(xLayer_a0Cuda);
    hipFree(yLayer_a1Cuda);
    hipFree(zLayer_a2Cuda);
    hipFree(xLayer_a3Cuda);
    hipFree(yLayer_a4Cuda);
    hipFree(zLayer_a5Cuda);
    hipFree(xLayer_a6Cuda);
    hipFree(yLayer_a6Cuda);
    hipFree(zLayer_a6Cuda);
    delete[] xLayer_a3;
    delete[] yLayer_a4;
    delete[] zLayer_a5;
    delete[] xLayer_a6;
    delete[] yLayer_a6;
    delete[] zLayer_a6;
}

void SampleSDFForward(Tensor &out, Tensor &sdfGrid, Tensor &pointList, Tensor &indexList)
{
    const unsigned int theardCount = pointList.size(0);
    const unsigned int blockSize = commonBlockSize;
    const unsigned int gridSize = GetGridSize(blockSize, theardCount);
    SampleSDFForwardKernel<<<gridSize, blockSize>>>(out.data<float>(), sdfGrid.data<float>(), (vec3f *)pointList.data<float>(),
                                                    indexList.data<int>(), theardCount);
}

void SampleNormalForward(Tensor &out, Tensor &sdfGrid, Tensor &pointList, Tensor &indexList)
{
    const unsigned int theardCount = pointList.size(0);
    const unsigned int blockSize = commonBlockSize;
    const unsigned int gridSize = GetGridSize(blockSize, theardCount);
    SampleNormalForwardKernel<<<gridSize, blockSize>>>((vec3f *)out.data<float>(), sdfGrid.data<float>(), (vec3f *)pointList.data<float>(), indexList.data<int>(), theardCount);
}